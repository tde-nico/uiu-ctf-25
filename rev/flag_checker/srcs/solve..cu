#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <map>



// https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html
#define CHECK(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t res, const char *func, const char *file, const int line)
{
	if (!res)
		return ;
	std::cerr << "CUDA error = " << static_cast<unsigned int>(res);
	std::cerr << " at " << file << ":" << line << " '" << func << "' \n";
	hipDeviceReset();
	exit(1);
}

typedef unsigned long long ull;

__device__ ull test_pt[] = {0x2265B1F5LL, 0x91B7584ALL, 0x0D8F16ADFLL, 0x0CD613E30LL, 0x0C386BBC4LL, 0x1027C4D1LL, 0x414C343CLL, 0x1E2FEB89LL};
__device__ ull test_ct[] = {0x0DC44BF5ELL, 0x5AFF1CECLL, 0x0E1E9B4C2LL, 0x1329B92LL, 0x8F9CA92ALL, 0x0E45C5B4LL, 0x604A4B91LL, 0x7081EB59LL};


__device__ ull F(ull a1, ull a2, ull a3)
{
  ull v5;
  ull v6;

  v5 = 1LL;
  v6 = a1 % a3;
  while ( a2 > 0 )
  {
    if ( (a2 & 1) != 0 )
      v5 = v6 * v5 % a3;
    v6 = v6 * v6 % a3;
    a2 >>= 1;
  }
  return v5;
}

__global__ void	brute() {
	ull exp = threadIdx.x + (blockIdx.x + (blockIdx.y + blockIdx.z * 256) * 256) * 256;
	for ( int i = 0; i <= 7; ++i ){
		if (F(test_pt[i], exp, 0xFFFFFF2FLL) == test_ct[i] ) {
			printf("Found: %d %016llx\n", i, exp);
		}
	}
}

int main(void)
{
	clock_t			start;
	clock_t			stop;

	dim3	blocks(256, 256, 256);
	dim3	threads(256);

	start = clock();

	brute<<<blocks, threads>>>();
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());

	stop = clock();
	std::cerr << "Took: " << ((double)(stop - start)) / CLOCKS_PER_SEC << "\n";

	return (0);
}
